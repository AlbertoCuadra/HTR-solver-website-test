#include "hip/hip_runtime.h"
// Copyright (c) "2019, by Stanford University
//               Developer: Mario Di Renzo
//               Affiliation: Center for Turbulence Research, Stanford University
//               URL: https://ctr.stanford.edu
//               Citation: Di Renzo, M., Lin, F., and Urzay, J. (2020).
//                         HTR solver: An open-source exascale-oriented task-based
//                         multi-GPU high-order code for hypersonic aerothermodynamics.
//                         Computer Physics Communications 255, 107262"
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//    * Redistributions of source code must retain the above copyright
//      notice, this list of conditions and the following disclaimer.
//    * Redistributions in binary form must reproduce the above copyright
//      notice, this list of conditions and the following disclaimer in the
//      documentation and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "prometeo_sensor.hpp"
#include "prometeo_sensor.inl"
#include "cuda_utils.hpp"

//-----------------------------------------------------------------------------
// KERNEL FOR UpdateDucrosSensorTask
//-----------------------------------------------------------------------------

__global__
void ComputeDucrosSensor_kernel(const AccessorWO<double, 3> DucrosSensor,
                                const AccessorRO<  Vec3, 3> velocity,
                                const AccessorRO<   int, 3> nType_csi,
                                const AccessorRO<   int, 3> nType_eta,
                                const AccessorRO<   int, 3> nType_zet,
                                const AccessorRO<double, 3> dcsi_d,
                                const AccessorRO<double, 3> deta_d,
                                const AccessorRO<double, 3> dzet_d,
                                const Rect<3> my_bounds,
                                const Rect<3> Fluid_bounds,
                                const double eps,
                                const coord_t size_x,
                                const coord_t size_y,
                                const coord_t size_z)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   int z = blockIdx.z * blockDim.z + threadIdx.z;

   if ((x < size_x) && (y < size_y) && (z < size_z)) {
      const Point<3> p = Point<3>(x + my_bounds.lo.x,
                                  y + my_bounds.lo.y,
                                  z + my_bounds.lo.z);
      DucrosSensor[p] = UpdateDucrosSensorTask::DucrosSensor(velocity,
                                     nType_csi, nType_eta, nType_zet,
                                     dcsi_d, deta_d, dzet_d,
                                     p, Fluid_bounds, eps);

   }
}

__host__
void UpdateDucrosSensorTask::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime)
{
   assert(regions.size() == 3);
   assert(futures.size() == 0);

   // Accessors for variables in the Ghost regions
   const AccessorRO<  Vec3, 3> acc_velocity         (regions[0], FID_velocity);

   // Accessors for metrics
   const AccessorRO<   int, 3> acc_nType_x          (regions[1], FID_nType_x);
   const AccessorRO<   int, 3> acc_nType_y          (regions[1], FID_nType_y);
   const AccessorRO<   int, 3> acc_nType_z          (regions[1], FID_nType_z);
   const AccessorRO<double, 3> acc_dcsi_d           (regions[1], FID_dcsi_d);
   const AccessorRO<double, 3> acc_deta_d           (regions[1], FID_deta_d);
   const AccessorRO<double, 3> acc_dzet_d           (regions[1], FID_dzet_d);

   // Accessors for shock sensor
   const AccessorWO<double, 3> acc_DucrosSensor     (regions[2], FID_DucrosSensor);

   // Extract execution domains
   Rect<3> r_MyFluid = runtime->get_index_space_domain(ctx, regions[1].get_logical_region().get_index_space());
   Rect<3> Fluid_bounds = args.Fluid_bounds;

   // Compute vorticity scale
   const double eps = max(args.vorticityScale*args.vorticityScale, 1e-6);

   // Launch the kernel to update the Ducros sensor
   const int threads_per_block = 256;
   const dim3 TPB_3d = splitThreadsPerBlock<Xdir>(threads_per_block, r_MyFluid);
   const dim3 num_blocks_3d = dim3((getSize<Xdir>(r_MyFluid) + (TPB_3d.x - 1)) / TPB_3d.x,
                                   (getSize<Ydir>(r_MyFluid) + (TPB_3d.y - 1)) / TPB_3d.y,
                                   (getSize<Zdir>(r_MyFluid) + (TPB_3d.z - 1)) / TPB_3d.z);
   ComputeDucrosSensor_kernel<<<num_blocks_3d, TPB_3d>>>(
                              acc_DucrosSensor, acc_velocity,
                              acc_nType_x, acc_nType_y, acc_nType_z,
                              acc_dcsi_d, acc_deta_d, acc_dzet_d,
                              r_MyFluid, Fluid_bounds, eps,
                              getSize<Xdir>(r_MyFluid), getSize<Ydir>(r_MyFluid), getSize<Zdir>(r_MyFluid));
}

//-----------------------------------------------------------------------------
// KERNEL FOR UpdateShockSensorTask
//-----------------------------------------------------------------------------

template<direction dir>
__global__
void UpdateShockSensor_kernel(const AccessorRO<double, 3> DucrosSensor,
                              const AccessorRO<VecNEq, 3> Conserved,
                              const AccessorRO<   int, 3> nType,
                              const AccessorWO<  bool, 3> shockSensor,
                              const Rect<3> my_bounds,
                              const Rect<3> Fluid_bounds,
                              const coord_t size_x,
                              const coord_t size_y,
                              const coord_t size_z,
                              const coord_t size)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   int z = blockIdx.z * blockDim.z + threadIdx.z;

   if ((x < size_x) && (y < size_y) && (z < size_z)) {
      const Point<3> p = Point<3>(x + my_bounds.lo.x,
                                  y + my_bounds.lo.y,
                                  z + my_bounds.lo.z);
      const Point<3> pM2 = warpPeriodic<dir, Minus>(Fluid_bounds, p, size, offM2(nType[p]));
      const Point<3> pM1 = warpPeriodic<dir, Minus>(Fluid_bounds, p, size, offM1(nType[p]));
      const Point<3> pP1 = warpPeriodic<dir, Plus >(Fluid_bounds, p, size, offP1(nType[p]));
      const Point<3> pP2 = warpPeriodic<dir, Plus >(Fluid_bounds, p, size, offP2(nType[p]));
      const Point<3> pP3 = warpPeriodic<dir, Plus >(Fluid_bounds, p, size, offP3(nType[p]));

      const double Phi = max(max(max(max(max(
                           DucrosSensor[pM2],
                           DucrosSensor[pM1]),
                           DucrosSensor[p  ]),
                           DucrosSensor[pP1]),
                           DucrosSensor[pP2]),
                           DucrosSensor[pP3]);

      bool sensor = true;
      #pragma unroll
      for (int i=0; i<nSpec; i++)
         sensor = sensor && TENOsensor::TENOA(Conserved[pM2][i], Conserved[pM1][i], Conserved[p  ][i],
                                              Conserved[pP1][i], Conserved[pP2][i], Conserved[pP3][i],
                                              nType[p], Phi);
      shockSensor[p] = sensor;
   }
}

template<direction dir>
__host__
void UpdateShockSensorTask<dir>::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime)
{
   assert(regions.size() == 3);
   assert(futures.size() == 0);

   // Accessors for variables in the Ghost regions
   const AccessorRO<VecNEq, 3> acc_Conserved        (regions[0], FID_Conserved);
   const AccessorRO<double, 3> acc_DucrosSensor     (regions[0], FID_DucrosSensor);

   // Accessors for node type
   const AccessorRO<   int, 3> acc_nType            (regions[1], FID_nType);

   // Accessors for shock sensor
   const AccessorWO<  bool, 3> acc_shockSensor      (regions[2], FID_shockSensor);

   // Extract execution domains
   Rect<3> r_MyFluid = runtime->get_index_space_domain(ctx, regions[2].get_logical_region().get_index_space());
   Rect<3> Fluid_bounds = args.Fluid_bounds;
   const coord_t size = getSize<dir>(Fluid_bounds);

   // Launch the kernel to update the shock sensor
   const int threads_per_block = 256;
   const dim3 TPB_3d = splitThreadsPerBlock<dir>(threads_per_block, r_MyFluid);
   const dim3 num_blocks_3d = dim3((getSize<Xdir>(r_MyFluid) + (TPB_3d.x - 1)) / TPB_3d.x,
                                   (getSize<Ydir>(r_MyFluid) + (TPB_3d.y - 1)) / TPB_3d.y,
                                   (getSize<Zdir>(r_MyFluid) + (TPB_3d.z - 1)) / TPB_3d.z);
   UpdateShockSensor_kernel<dir><<<num_blocks_3d, TPB_3d>>>(
                              acc_DucrosSensor, acc_Conserved,
                              acc_nType, acc_shockSensor,
                              r_MyFluid, Fluid_bounds,
                              getSize<Xdir>(r_MyFluid), getSize<Ydir>(r_MyFluid), getSize<Zdir>(r_MyFluid), size);
}

template void UpdateShockSensorTask<Xdir>::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime);

template void UpdateShockSensorTask<Ydir>::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime);

template void UpdateShockSensorTask<Zdir>::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime);

