#include "hip/hip_runtime.h"
// Copyright (c) "2019, by Stanford University
//               Developer: Mario Di Renzo
//               Affiliation: Center for Turbulence Research, Stanford University
//               URL: https://ctr.stanford.edu
//               Citation: Di Renzo, M., Lin, F., and Urzay, J. (2020).
//                         HTR solver: An open-source exascale-oriented task-based
//                         multi-GPU high-order code for hypersonic aerothermodynamics.
//                         Computer Physics Communications 255, 107262"
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//    * Redistributions of source code must retain the above copyright
//      notice, this list of conditions and the following disclaimer.
//    * Redistributions in binary form must reproduce the above copyright
//      notice, this list of conditions and the following disclaimer in the
//      documentation and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "prometeo_variables.hpp"
#include "cuda_utils.hpp"

// Declare a constant memory that will hold the Mixture struct (initialized in prometeo_mixture.cu)
extern __device__ __constant__ Mix mix;

//-----------------------------------------------------------------------------
// KERNELS FOR UpdatePropertiesFromPrimitiveTask
//-----------------------------------------------------------------------------

__global__
void UpdatePropertiesFromPrimitive_kernel(const AccessorRO<double, 3> pressure,
                                          const AccessorRO<double, 3> temperature,
                                          const AccessorRO<VecNSp, 3> MolarFracs,
                                          const AccessorRO<  Vec3, 3> velocity,
                                          const AccessorWO<VecNSp, 3> MassFracs,
                                          const AccessorWO<double, 3> rho,
                                          const AccessorWO<double, 3> mu,
                                          const AccessorWO<double, 3> lam,
                                          const AccessorWO<VecNSp, 3> Di,
                                          const AccessorWO<double, 3> SoS,
#if (defined(ELECTRIC_FIELD) && (nIons > 0))
                                          const AccessorWO<VecNIo, 3> Ki,
#endif
                                          const Rect<3> my_bounds,
                                          const coord_t  size_x,
                                          const coord_t  size_y,
                                          const coord_t  size_z)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   int z = blockIdx.z * blockDim.z + threadIdx.z;

   if ((x < size_x) && (y < size_y) && (z < size_z)) {
      const Point<3> p = Point<3>(x + my_bounds.lo.x,
                                  y + my_bounds.lo.y,
                                  z + my_bounds.lo.z);
      // Mixture check
      assert(mix.CheckMixture(MolarFracs[p]));
      UpdatePropertiesFromPrimitiveTask::UpdateProperties(
                       pressure, temperature, MolarFracs, velocity,
                       MassFracs,
                       rho, mu, lam, Di, SoS,
#if (defined(ELECTRIC_FIELD) && (nIons > 0))
                       Ki,
#endif
                       p, mix);
   }
}

__host__
void UpdatePropertiesFromPrimitiveTask::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime)
{
   assert(regions.size() == 2);
   assert(futures.size() == 0);

   // Accessors for primitive variables
   const AccessorRO<double, 3> acc_pressure         (regions[0], FID_pressure);
   const AccessorRO<double, 3> acc_temperature      (regions[0], FID_temperature);
   const AccessorRO<VecNSp, 3> acc_MolarFracs       (regions[0], FID_MolarFracs);
   const AccessorRO<  Vec3, 3> acc_velocity         (regions[0], FID_velocity);

   const AccessorWO<VecNSp, 3> acc_MassFracs        (regions[1], FID_MassFracs);

   // Accessors for properties
   const AccessorWO<double, 3> acc_rho              (regions[1], FID_rho);
   const AccessorWO<double, 3> acc_mu               (regions[1], FID_mu);
   const AccessorWO<double, 3> acc_lam              (regions[1], FID_lam);
   const AccessorWO<VecNSp, 3> acc_Di               (regions[1], FID_Di);
   const AccessorWO<double, 3> acc_SoS              (regions[1], FID_SoS);
#if (defined(ELECTRIC_FIELD) && (nIons > 0))
   const AccessorWO<VecNIo, 3> acc_Ki               (regions[1], FID_Ki);
#endif

   // Extract execution domains
   Rect<3> r_Fluid = runtime->get_index_space_domain(ctx, regions[1].get_logical_region().get_index_space());

   // Launch the kernel
   const int threads_per_block = 256;
   const dim3 TPB_3d = splitThreadsPerBlock<Xdir>(threads_per_block, r_Fluid);
   const dim3 num_blocks_3d = dim3((getSize<Xdir>(r_Fluid) + (TPB_3d.x - 1)) / TPB_3d.x,
                                   (getSize<Ydir>(r_Fluid) + (TPB_3d.y - 1)) / TPB_3d.y,
                                   (getSize<Zdir>(r_Fluid) + (TPB_3d.z - 1)) / TPB_3d.z);
   UpdatePropertiesFromPrimitive_kernel<<<num_blocks_3d, TPB_3d>>>(
                        acc_pressure, acc_temperature, acc_MolarFracs,
                        acc_velocity, acc_MassFracs,
                        acc_rho, acc_mu, acc_lam, acc_Di, acc_SoS,
#if (defined(ELECTRIC_FIELD) && (nIons > 0))
                        acc_Ki,
#endif
                        r_Fluid, getSize<Xdir>(r_Fluid), getSize<Ydir>(r_Fluid), getSize<Zdir>(r_Fluid));
}

//-----------------------------------------------------------------------------
// KERNELS FOR UpdateConservedFromPrimitiveTask
//-----------------------------------------------------------------------------

__global__
void UpdateConservedFromPrimitive_kernel(const AccessorRO<VecNSp, 3> MassFracs,
                                         const AccessorRO<double, 3> temperature,
                                         const AccessorRO<  Vec3, 3> velocity,
                                         const AccessorRO<double, 3> rho,
                                         const AccessorWO<VecNEq, 3> Conserved,
                                         const Rect<3> my_bounds,
                                         const coord_t  size_x,
                                         const coord_t  size_y,
                                         const coord_t  size_z)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   int z = blockIdx.z * blockDim.z + threadIdx.z;

   if ((x < size_x) && (y < size_y) && (z < size_z)) {
      const Point<3> p = Point<3>(x + my_bounds.lo.x,
                                  y + my_bounds.lo.y,
                                  z + my_bounds.lo.z);
      // Mixture check
      assert(mix.CheckMixture(MassFracs[p]));
      UpdateConservedFromPrimitiveTask::UpdateConserved(
                     MassFracs, temperature, velocity,
                     rho, Conserved,
                     p, mix);
   }
}

__host__
void UpdateConservedFromPrimitiveTask::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime)
{
   assert(regions.size() == 2);
   assert(futures.size() == 0);

   // Accessors for primitive variables
   const AccessorRO<VecNSp, 3> acc_MassFracs        (regions[0], FID_MassFracs);
   const AccessorRO<double, 3> acc_temperature      (regions[0], FID_temperature);
   const AccessorRO<  Vec3, 3> acc_velocity         (regions[0], FID_velocity);

   // Accessors for properties
   const AccessorRO<double, 3> acc_rho              (regions[0], FID_rho);

   // Accessors for conserved variables
   const AccessorWO<VecNEq, 3> acc_Conserved        (regions[1], FID_Conserved);

   // Extract execution domains
   Domain r_Fluid = runtime->get_index_space_domain(ctx, regions[1].get_logical_region().get_index_space());

   // Launch the kernel (launch domain might be composed by multiple rectangles)
   for (RectInDomainIterator<3> Rit(r_Fluid); Rit(); Rit++) {
      const int threads_per_block = 256;
      const dim3 TPB_3d = splitThreadsPerBlock<Xdir>(threads_per_block, (*Rit));
      const dim3 num_blocks_3d = dim3((getSize<Xdir>(*Rit) + (TPB_3d.x - 1)) / TPB_3d.x,
                                      (getSize<Ydir>(*Rit) + (TPB_3d.y - 1)) / TPB_3d.y,
                                      (getSize<Zdir>(*Rit) + (TPB_3d.z - 1)) / TPB_3d.z);
      UpdateConservedFromPrimitive_kernel<<<num_blocks_3d, TPB_3d>>>(
                           acc_MassFracs, acc_temperature, acc_velocity,
                           acc_rho, acc_Conserved, (*Rit),
                           getSize<Xdir>(*Rit), getSize<Ydir>(*Rit), getSize<Zdir>(*Rit));
   }
}

//-----------------------------------------------------------------------------
// KERNELS FOR UpdatePrimitiveFromConservedTask
//-----------------------------------------------------------------------------

__global__
void UpdatePrimitiveFromConserved_kernel(const AccessorRO<VecNEq, 3> Conserved,
                                         const AccessorRW<double, 3> temperature,
                                         const AccessorWO<double, 3> pressure,
                                         const AccessorWO<VecNSp, 3> MolarFracs,
                                         const AccessorWO<  Vec3, 3> velocity,
                                         const Rect<3> my_bounds,
                                         const coord_t  size_x,
                                         const coord_t  size_y,
                                         const coord_t  size_z)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   int z = blockIdx.z * blockDim.z + threadIdx.z;

   if ((x < size_x) && (y < size_y) && (z < size_z)) {
      const Point<3> p = Point<3>(x + my_bounds.lo.x,
                                  y + my_bounds.lo.y,
                                  z + my_bounds.lo.z);
      UpdatePrimitiveFromConservedTask::UpdatePrimitive(
                     Conserved, temperature, pressure,
                     MolarFracs, velocity,
                     p, mix);
   }
}

__host__
void UpdatePrimitiveFromConservedTask::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime)
{
   assert(regions.size() == 2);
   assert(futures.size() == 0);

   // Accessors for conserved variables
   const AccessorRO<VecNEq, 3> acc_Conserved        (regions[0], FID_Conserved);

   // Accessors for temperature variables
   const AccessorRW<double, 3> acc_temperature      (regions[1], FID_temperature);

   // Accessors for primitive variables
   const AccessorWO<double, 3> acc_pressure         (regions[1], FID_pressure);
   const AccessorWO<VecNSp, 3> acc_MolarFracs       (regions[1], FID_MolarFracs);
   const AccessorWO<  Vec3, 3> acc_velocity         (regions[1], FID_velocity);

   // Extract execution domains
   Rect<3> r_Fluid = runtime->get_index_space_domain(ctx, regions[1].get_logical_region().get_index_space());

   // Launch the kernel
   const int threads_per_block = 256;
   const dim3 TPB_3d = splitThreadsPerBlock<Xdir>(threads_per_block, r_Fluid);
   const dim3 num_blocks_3d = dim3((getSize<Xdir>(r_Fluid) + (TPB_3d.x - 1)) / TPB_3d.x,
                                   (getSize<Ydir>(r_Fluid) + (TPB_3d.y - 1)) / TPB_3d.y,
                                   (getSize<Zdir>(r_Fluid) + (TPB_3d.z - 1)) / TPB_3d.z);
   UpdatePrimitiveFromConserved_kernel<<<num_blocks_3d, TPB_3d>>>(
                        acc_Conserved, acc_temperature, acc_pressure,
                        acc_MolarFracs, acc_velocity, r_Fluid,
                        getSize<Xdir>(r_Fluid), getSize<Ydir>(r_Fluid), getSize<Zdir>(r_Fluid));
}

