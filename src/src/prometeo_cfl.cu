#include "hip/hip_runtime.h"
// Copyright (c) "2019, by Stanford University
//               Developer: Mario Di Renzo
//               Affiliation: Center for Turbulence Research, Stanford University
//               URL: https://ctr.stanford.edu
//               Citation: Di Renzo, M., Lin, F., and Urzay, J. (2020).
//                         HTR solver: An open-source exascale-oriented task-based
//                         multi-GPU high-order code for hypersonic aerothermodynamics.
//                         Computer Physics Communications 255, 107262"
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//    * Redistributions of source code must retain the above copyright
//      notice, this list of conditions and the following disclaimer.
//    * Redistributions in binary form must reproduce the above copyright
//      notice, this list of conditions and the following disclaimer in the
//      documentation and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "prometeo_cfl.hpp"
#include "cuda_utils.hpp"

// Declare a constant memory that will hold the Mixture struct (initialized in prometeo_mixture.cu)
extern __device__ __constant__ Mix mix;

//-----------------------------------------------------------------------------
// KERNEL FOR CalculateMaxSpectralRadiusTask
//-----------------------------------------------------------------------------

__global__
void CalculateMaxSpectralRadius_kernel(const DeferredBuffer<double, 1> buffer,
                                       const AccessorRO<double, 3> dcsi,
                                       const AccessorRO<double, 3> deta,
                                       const AccessorRO<double, 3> dzet,
                                       const AccessorRO<double, 3> temperature,
                                       const AccessorRO<VecNSp, 3> MassFracs,
                                       const AccessorRO<  Vec3, 3> velocity,
                                       const AccessorRO<double, 3> rho,
                                       const AccessorRO<double, 3> mu,
                                       const AccessorRO<double, 3> lam,
                                       const AccessorRO<VecNSp, 3> Di,
                                       const AccessorRO<double, 3> SoS,
#if (defined(ELECTRIC_FIELD) && (nIons > 0))
                                       const AccessorRO<VecNIo, 3> Ki,
                                       const AccessorRO<  Vec3, 3> eField,
#endif
                                       const Rect<3> my_bounds,
                                       const coord_t size_x,
                                       const coord_t size_y,
                                       const coord_t size_z)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   int z = blockIdx.z * blockDim.z + threadIdx.z;

   double my_r = 0.0; // Spectral radius cannot be lower than 0
   if ((x < size_x) && (y < size_y) && (z < size_z)) {
      const Point<3> p = Point<3>(x + my_bounds.lo.x,
                                  y + my_bounds.lo.y,
                                  z + my_bounds.lo.z);
      my_r = CalculateMaxSpectralRadiusTask::CalculateMaxSpectralRadius(
                                    dcsi, deta, dzet,
                                    temperature, MassFracs, velocity,
                                    rho, mu, lam, Di, SoS,
#if (defined(ELECTRIC_FIELD) && (nIons > 0))
                                    Ki, eField,
#endif
                                    p, mix);
   }
   reduceMax(my_r, buffer);
}

__host__
DeferredValue<double> CalculateMaxSpectralRadiusTask::gpu_base_impl(
                      const Args &args,
                      const std::vector<PhysicalRegion> &regions,
                      const std::vector<Future>         &futures,
                      Context ctx, Runtime *runtime)
{
   assert(regions.size() == 1);
   assert(futures.size() == 0);

   // Accessor for metrics
   const AccessorRO<double, 3> acc_dcsi_d           (regions[0], FID_dcsi_d);
   const AccessorRO<double, 3> acc_deta_d           (regions[0], FID_deta_d);
   const AccessorRO<double, 3> acc_dzet_d           (regions[0], FID_dzet_d);

   // Accessors for primitive variables
   const AccessorRO<VecNSp, 3> acc_MassFracs        (regions[0], FID_MassFracs);
   const AccessorRO<double, 3> acc_temperature      (regions[0], FID_temperature);
   const AccessorRO<  Vec3, 3> acc_velocity         (regions[0], FID_velocity);

   // Accessors for properties
   const AccessorRO<double, 3> acc_rho              (regions[0], FID_rho);
   const AccessorRO<double, 3> acc_mu               (regions[0], FID_mu);
   const AccessorRO<double, 3> acc_lam              (regions[0], FID_lam);
   const AccessorRO<VecNSp, 3> acc_Di               (regions[0], FID_Di);
   const AccessorRO<double, 3> acc_SoS              (regions[0], FID_SoS);
#if (defined(ELECTRIC_FIELD) && (nIons > 0))
   const AccessorRO<VecNIo, 3> acc_Ki               (regions[0], FID_Ki);

   // Accessors for primitive variables
   const AccessorRO<  Vec3, 3> acc_eField           (regions[0], FID_electricField);
#endif

   // Extract execution domains
   Rect<3> r_MyFluid = runtime->get_index_space_domain(ctx, regions[0].get_logical_region().get_index_space());

   // Define thread grid
   const int threads_per_block = 256;
   dim3 TPB_3d = splitThreadsPerBlock<Xdir>(threads_per_block, r_MyFluid);
   while (TPB_3d.x*TPB_3d.y*TPB_3d.z < 32) TPB_3d.x++;
   const dim3 num_blocks_3d = dim3((getSize<Xdir>(r_MyFluid) + (TPB_3d.x - 1)) / TPB_3d.x,
                                   (getSize<Ydir>(r_MyFluid) + (TPB_3d.y - 1)) / TPB_3d.y,
                                   (getSize<Zdir>(r_MyFluid) + (TPB_3d.z - 1)) / TPB_3d.z);

   // Store the maximum value per block in a deferred buffer
   const size_t total_blocks = num_blocks_3d.x*num_blocks_3d.y*num_blocks_3d.z;
   const Rect<1> bounds(Point<1>(0), Point<1>(total_blocks - 1));
   DeferredBuffer<double, 1> buffer(bounds, Memory::GPU_FB_MEM);
   CalculateMaxSpectralRadius_kernel<<<num_blocks_3d, TPB_3d>>>(buffer,
                           acc_dcsi_d, acc_deta_d, acc_dzet_d,
                           acc_temperature, acc_MassFracs, acc_velocity,
                           acc_rho, acc_mu, acc_lam, acc_Di, acc_SoS,
#if (defined(ELECTRIC_FIELD) && (nIons > 0))
                           acc_Ki, acc_eField,
#endif
                           r_MyFluid, getSize<Xdir>(r_MyFluid), getSize<Ydir>(r_MyFluid), getSize<Zdir>(r_MyFluid));

   // Reduce spectral radii into r
   DeferredValue<double> r(0.0);

   // We use at most 1024 blocks
   dim3 TPB((total_blocks > 1024) ? 1024 : total_blocks, 1, 1);
   // Round up to the nearest multiple of warps
   while ((TPB.x % 32) != 0) TPB.x++;
   const dim3 num_blocks(1, 1, 1);
   ReduceBufferMax_kernel<<<num_blocks, TPB>>>(buffer, r, total_blocks);

   return r;
}

